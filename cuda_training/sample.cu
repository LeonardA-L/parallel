// http://www.nvidia.com/docs/io/116711/sc11-cuda-c-basics.pdf
/* C stuff */
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include <errno.h>

/* Cuda stuff */
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define	N	(2048*2048)
#define TH_PER_BLOCK	512

__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void randomints(int* a){
   int i;
   for (i = 0; i < N; ++i)
    a[i] = 1;
}

int main(void) {
	int *a,*b,*c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);
	
	hipMalloc((void**) &d_a, size);
	hipMalloc((void**) &d_b, size);
	hipMalloc((void**) &d_c, size);
	
	a = (int*)malloc(size); randomints(a);
	b = (int*)malloc(size); randomints(b);
	c = (int*)malloc(size);

	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N/TH_PER_BLOCK,TH_PER_BLOCK>>>(d_a, d_b, d_c);
	
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	printf("It is %d\n", c[0]);
	
	free(a); free(b); free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}
