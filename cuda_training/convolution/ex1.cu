#include "hip/hip_runtime.h"

/*****************************************************************************
CUDA PROGRAMMING EXAMPLES

Authors: Christian Wolf, LIRIS, CNRS, INSA-Lyon 
christian.wolf@liris.cnrs.fr

Changelog:
03.07.15 cw: begin development
*****************************************************************************/

/* C stuff */
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include <errno.h>

// Open-CV for the vision stuff
#include <opencv2/opencv.hpp>

/* Cuda stuff */
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

using namespace std;

#define blockSize	16
#define TILE_WIDTH	blockSize

clock_t LastProfilingClock=clock();

/***************************************************************************
 Writes profiling output (milli-seconds since last call)
 ***************************************************************************/

extern clock_t LastProfilingClock;

inline float profiling (const char *s, clock_t *whichClock=NULL) 
{
	if (whichClock==NULL)
		whichClock=&LastProfilingClock;

    clock_t newClock=clock();
    float res = (float) (newClock-*whichClock) / (float) CLOCKS_PER_SEC;
    if (s!=NULL)
        std::cerr << "Time: " << s << ": " << res << std::endl; 
    *whichClock = newClock;
    return res;
}

inline float profilingTime (const char *s, time_t *whichClock) 
{
    time_t newTime=time(NULL);
    float res = (float) (newTime-*whichClock);
    if (s!=NULL)
        std::cerr << "Time(real): " << s << ": " << res << std::endl; 
    return res;
}

/* Our stuff */

/***************************************************************************
 USAGE
 ***************************************************************************/

void usage (char *com) 
{
    std::cerr<< "usage: " << com << " <imagename>\n";
    exit(1);
}

/***************************************************************************
 The CPU version
 ***************************************************************************/

void cpuFilter(unsigned char *in, unsigned char * resarr, int rows, int cols){
	for (int y=1; y<rows-1; ++y){
		for (int x=1; x<cols-1; ++x){
			//cout << (int)imarr[x*rows+y] << endl;
			/*int total = 0;
			if(y > 0){	// !TOP
				if(x > 0){	// !LEFT
					total += imarr[(x-1)*rows+(y-1)] * 1;
				}
				total += imarr[(x)*rows+(y-1)] * 2;
				if(x < cols -1){	// !RIGHT
					total += imarr[(x+1)*rows+(y-1)] * 1;
				}
			}
				if(x > 0){	// !LEFT
					total += imarr[(x-1)*rows+(y)] * 2;
				}
				total += imarr[(x)*rows+(y)] * 4;
				if(x < cols -1){	// !RIGHT
					total += imarr[(x+1)*rows+(y)] * 2;
				}
			if(y < rows - 1){
				if(x > 0){	// !LEFT
					total += imarr[(x-1)*rows+(y+1)] * 1;
				}
				total += imarr[(x)*rows+(y+1)] * 2;
				if(x < cols -1){	// !RIGHT
					total += imarr[(x+1)*rows+(y+1)] * 1;
				}
			}
			total /= 16;*/
			int total = (
            4.0*in[x*rows+y] +
            2.0*in[(x-1)*rows+y] +
            2.0*in[(x+2)*rows+y] +
            2.0*in[x*rows+y+1] +
            2.0*in[x*rows+y-1] +
            in[(x-1)*rows+y-1] +
            in[(x-1)*rows+y+1] +
            in[(x+1)*rows+y-1] +
            in[(x+1)*rows+y+1]
            )/16.0;
			
			if(total < 0) total = 0;
			if(total > 255) total = 255;
			//cout << total << endl;
			resarr[x*rows+y] = (unsigned char)total;
		}
	}
}

/***************************************************************************
 The GPU version - the kernel
 ***************************************************************************/

__global__ void onePixel(unsigned char *in, unsigned char *resarr, int * d_rows) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int rows = *d_rows;
		
		int shx = threadIdx.x;
		int shy = threadIdx.y;
		
		__shared__ unsigned char shIn[TILE_WIDTH * TILE_WIDTH];
		
		shIn[threadIdx.y * blockSize +	threadIdx.x] = in[x*rows+y];
		
		__syncthreads();
		
		int total;
		if(shx > 0 && shy > 0 && shx < blockSize-1 && shy < blockSize-1){
			total = (
			4.0*shIn[shx+shy*blockSize] +
			2.0*shIn[(shx-1)+shy*blockSize] +
			2.0*shIn[(shx+2)+shy*blockSize] +
			2.0*shIn[shx+(shy+1)*blockSize] +
			2.0*shIn[shx+(shy-1)*blockSize] +
			shIn[(shx-1)+(shy-1)*blockSize] +
			shIn[(shx-1)+(shy+1)*blockSize] +
			shIn[(shx+1)+(shy-1)*blockSize] +
			shIn[(shx+1)+(shy+1)*blockSize]
			)/16.0;
			//total = 255;
		}
		else{
			// Non shared memory
			total = (
			4.0*in[x*rows+y] +
			2.0*in[(x-1)*rows+y] +
			2.0*in[(x+2)*rows+y] +
			2.0*in[x*rows+y+1] +
			2.0*in[x*rows+y-1] +
			in[(x-1)*rows+y-1] +
			in[(x-1)*rows+y+1] +
			in[(x+1)*rows+y-1] +
			in[(x+1)*rows+y+1]
			)/16.0;
		}
		
		if(total < 0) total = 0;
		if(total > 255) total = 255;
		//cout << total << endl;
		resarr[x*rows+y] = (unsigned char)total;
}

 /***************************************************************************
 The GPU version - the host code
 ***************************************************************************/

void testError(int ok, char* message){
	if(ok != hipSuccess){
		cerr << message << endl;
	}
}

void gpuFilter(unsigned char *in, unsigned char * resarr, int rows, int cols){
	long size = sizeof(unsigned char)*cols*rows;
	unsigned char *d_in, *d_out;
	int* d_rows;
	
	hipError_t ok;
	
	ok=hipMalloc((void**) &d_in, size);
	testError(ok, "hipMalloc 1 error");
	ok=hipMalloc((void**) &d_out, size);
	testError(ok, "hipMalloc 2 error");
	ok=hipMalloc((void**) &d_rows, sizeof(int));
	testError(ok, "hipMalloc 3 error");
	
	ok=hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	testError(ok, "hipMemcpy 1 error");
	ok=hipMemcpy(d_rows, &rows, sizeof(int), hipMemcpyHostToDevice);
	testError(ok, "hipMemcpy 2 error");
	
	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(rows/blockSize, cols/blockSize);
	
	/*cout << dimBlock.x << " " << dimBlock.y << endl;
	cout << dimGrid.x << " " << dimGrid.y << endl;*/
	
	onePixel<<<dimGrid, dimBlock>>>(d_in, d_out, d_rows);
	ok = hipGetLastError();
	cerr << "CUDA Status :"<< hipGetErrorString(ok) << endl;
	testError(ok, "error kernel launch");
	
	//cout << &resarr << endl;
	
	ok=hipMemcpy(resarr, d_out, size, hipMemcpyDeviceToHost);
	testError(ok, "hipMemcpy deviceToHost error");
	
	ok=hipFree(d_in);
	testError(ok, "hipFree 1 error");
	ok=hipFree(d_out);
	testError(ok, "hipFree 2 error");
	ok=hipFree(d_rows);
	testError(ok, "hipFree 3 error");
	
}
	

/***************************************************************************
 Main program
 ***************************************************************************/


int main (int argc, char **argv)
{
	int c;
	// Argument processing
    while ((c =	getopt (argc, argv,	"h")) != EOF) 
    {
		switch (c) {

			case 'h':
				usage(*argv);
				break;
	
			case '?':
				usage (*argv);
				std::cerr << "\n" << "*** Problem parsing the options!\n\n";
				exit (1);
		}
	}	

    int requiredArgs=2;

	if (argc-optind!=requiredArgs) 
    {
        usage (*argv);
		exit (1);
	}
	char *inputfname=argv[optind];
	char *outputfname=argv[optind+1];

	cv::Mat im = cv::imread(inputfname,-1);
	if (!im.data)
	{
		std::cerr << "*** Cannot load image: " << inputfname << "\n";
		exit(1);
	}
	std::cout << "=====================================================\n"
		<< "Loaded image of size " << im.cols << "x" << im.rows << ".\n";
	cv::Mat result (im.rows, im.cols, CV_8U);

	// Copy the cv::Mat into a linear array
	unsigned char *imarr = new unsigned char [im.cols*im.rows];
	for (int y=0; y<im.rows; ++y)
	for (int x=0; x<im.cols; ++x)
		imarr[x*im.rows+y] = im.at<unsigned char>(y,x);
	unsigned char *resarr = new unsigned char [im.cols*im.rows];
	profiling (NULL);
	int nMax = 1000;

	// Each version is run a 100 times to have 
	// a better idea on run time
	/*
	for (int i=0; i<nMax; ++i)
		cpuFilter(imarr, resarr, im.rows, im.cols);

	profiling ("CPU version");
	*/
	for (int i=0; i<nMax; ++i)
		gpuFilter(imarr, resarr, im.rows, im.cols);

	profiling ("GPU version");

	// Copy the linear array back to the cv::Mat
	for (int y=0; y<im.rows; ++y)
	for (int x=0; x<im.cols; ++x)
		result.at<unsigned char>(y,x) = resarr[x*im.rows+y];

	imwrite (outputfname, result);

    std::cout << "Program terminated correctly.\n";
    return 0;
}

